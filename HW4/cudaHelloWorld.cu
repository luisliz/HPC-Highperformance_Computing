
#include <hip/hip_runtime.h>
#include <stdio.h> 

const int N = 16; 
const int blocksize = 16; 

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void hello(char *a, int *b) {
				a[threadIdx.x] += b[threadIdx.x]; 
}

int main() {
				char a[N] = "Hello \n\n\n\n\n\n"; 
				int b[N] = {15, 10, 6, 0, -11, 1, 0, 0, 0, 0, 0,0,0,0,0,0}; 

				char *ad; 
				int *bd; 

				const int csize= N*sizeof(char); 
				const int isize = N*sizeof(int); 

				printf("%s", a); 

				gpuErrchk(hipMalloc((void**) &ad, csize));
				gpuErrchk(hipMalloc((void**) &bd, isize));
				gpuErrchk(hipMemcpy(ad, a, csize, hipMemcpyHostToDevice));
				gpuErrchk(hipMemcpy(bd, b, isize, hipMemcpyHostToDevice));

				dim3 dimBlock(blocksize, 1); 
				dim3 dimGrid(1, 1); 

				hello<<<dimGrid, dimBlock>>>(ad, bd); 
				gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );
				gpuErrchk(hipMemcpy(a, ad, csize, hipMemcpyHostToDevice)); 
				gpuErrchk(hipFree(ad)); 
				gpuErrchk(hipFree(bd));

				printf("%s\n", a); 
				return EXIT_SUCCESS; 
}

