
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

#define THREADS 1024
#define BLOCKS 65536
#define NUM_VALS THREADS*BLOCKS
#define ASCENDING 1
#define DESCENDING 0

void rand_nums(int *values, unsigned long length) {
	int i;
	for (i = 0; i < length; ++i) {
		values[i] = rand() % INT_MAX + 1;;
	}
}

void compAndSwap(int *a, int i, int j, int dir)
{
    int temp;
    if (dir==(a[i]>a[j])) {
        temp = a[j];
        a[j] = a[i];
        a[i] = temp;
        //swap(a[i], a[j]);
    }
}

void bitonicMerge(int *a, int low, int cnt, int dir) {
    if (cnt > 1) {
        int k = cnt / 2;
        for (int i = low; i < low + k; i++)
            compAndSwap(a, i, i + k, dir);
        bitonicMerge(a, low, k, dir);
        bitonicMerge(a, low + k, k, dir);
    }
}

void bitonicSort(int *values, int low, unsigned long n, int dir) {
    if(n>1) {
        int k = n/2;

        bitonicSort(values, low, k, ASCENDING);
        bitonicSort(values, low+k, k, DESCENDING);

        bitonicMerge(values, low, n, dir);
    }

}

int main(int argc, char *argv[]) {
    unsigned long n;
    double time_spent;
    clock_t begin, end;

    int k = 10;
	if(argc==2)
		k = atoi(argv[1]);

	n = pow(2,k);

	int *values = (int *) malloc(NUM_VALS * sizeof(int));
	int *origValues = (int *) malloc(NUM_VALS * sizeof(int));

	printf("\nk = %d, n = %ld\n", k, n); 

	rand_nums(values, n);

	for (unsigned long i = 0; i < n; i++)
		origValues[i] = values[i];

	time_spent = 0.0; 
	begin = clock(); 
	bitonicSort(values, 0, n, ASCENDING);
	end = clock();
	time_spent += (double)(end-begin) / CLOCKS_PER_SEC; 
	printf("\tElapsed time: %f seconds\n", time_spent); 
	
	free(values); 
	free(origValues); 
}




