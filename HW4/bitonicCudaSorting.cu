
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

#define THREADS 1024
#define BLOCKS 65536
#define NUM_VALS THREADS*BLOCKS

void rand_nums(int *values, int length) {
				int i;
				for (i = 0; i < length; ++i) {
								values[i] = rand() % INT_MAX + 1;;
				}
}


__global__ void bitonicMinorSort(int *innerValues, int j, int k) {
				unsigned int i, ixj;
				i = threadIdx.x + blockDim.x * blockIdx.x;

				ixj = i ^ j;

				if ((ixj) > i) {
								if ((i & k) == 0) {
												if (innerValues[i] > innerValues[ixj]) {
																int temp = innerValues[i];
																innerValues[i] = innerValues[ixj];
																innerValues[ixj] = temp;
												}
								}


								if ((i & k) != 0) {
												if (innerValues[i] < innerValues[ixj]) {
																int temp = innerValues[i];
																innerValues[i] = innerValues[ixj];
																innerValues[ixj] = temp;
												}
								}
				}
}

void bitonicSort(int *values, int n) {
				int *innerValues;

				size_t size = n * sizeof(int);

				hipMalloc((void **) &innerValues, size);
				hipMemcpy(innerValues, values, size, hipMemcpyHostToDevice);

				int blockSize;
				if(n < THREADS) {
								blockSize = 1;
				} else {
								blockSize = ceil(n/THREADS);
				}

				printf("%d BLOCKS", blockSize);
				dim3 blocks(blockSize, 1);
				dim3 threads(THREADS, 1);



				int j, k;

				for (k = 2; k <= n; k <<= 1) {
								for (j = k >> 1; j > 0; j = j >> 1) {
												bitonicMinorSort<<<blocks, threads>>> (innerValues, j, k);
								}
				}

				hipMemcpy(values, innerValues, size, hipMemcpyDeviceToHost);
				hipFree(innerValues);
}

int main(int argc, char *argv[]) {
				int *values = (int *) malloc(NUM_VALS * sizeof(int));
				int *origValues = (int *) malloc(NUM_VALS * sizeof(int));
				int k = 10;  
				if(argc==2) {
								k = atoi(argv[1]); 
								printf("%d\n", k); 

				} else {
								printf("Please input a correct value for k. Default k = 10 will be used.\n"); 
				}

				int n = pow(2,k); 
				printf("%d\n", n); 
				rand_nums(values, n);

				for (int i = 0; i < n; i++) {
								origValues[i] = values[i];
				}
				double time_spent = 0.0; 
				clock_t begin = clock(); 

				bitonicSort(values, n);
				hipDeviceSynchronize();
				clock_t end = clock();

				time_spent += (double)(end-begin) / CLOCKS_PER_SEC; 
				printf("Elapsed time is %f seconds", time_spent); 
				//printf("\nAfter:\n ");
				//for (int i = 0; i < n; i++) {
				//    printf("%d\t|%d\n ", values[i], origValues[i]);
				//}
				//printf("\n");
}



