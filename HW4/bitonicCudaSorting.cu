
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

#define THREADS 1024
#define BLOCKS 65536
#define NUM_VALS THREADS*BLOCKS

void rand_nums(int *values, int length) {
    int i;
    for (i = 0; i < length; ++i) {
        values[i] = rand() % INT_MAX + 1;;
    }
}


__global__ void bitonicMinorSort(int *innerValues, int j, int k) {
    unsigned int i, ixj;
    i = threadIdx.x + blockDim.x * blockIdx.x;

    ixj = i ^ j;

    if ((ixj) > i) {
        if ((i & k) == 0) {
            if (innerValues[i] > innerValues[ixj]) {
                int temp = innerValues[i];
                innerValues[i] = innerValues[ixj];
                innerValues[ixj] = temp;
            }
        }


        if ((i & k) != 0) {
            if (innerValues[i] < innerValues[ixj]) {
                int temp = innerValues[i];
                innerValues[i] = innerValues[ixj];
                innerValues[ixj] = temp;
            }
        }
    }
}

void bitonicSort(int *values, int n) {
    int *innerValues;

    size_t size = n * sizeof(int);

    hipMalloc((void **) &innerValues, size);
    hipMemcpy(innerValues, values, size, hipMemcpyHostToDevice);

    int blockSize;
    if(n < THREADS) {
         blockSize = 1;
    } else {
        blockSize = ceil(n/THREADS);
    }

    printf("%d BLOCKS", blockSize);
    dim3 blocks(blockSize, 1);
    dim3 threads(THREADS, 1);



    int j, k;

    for (k = 2; k <= n; k <<= 1) {
        for (j = k >> 1; j > 0; j = j >> 1) {
            bitonicMinorSort<<<blocks, threads>>> (innerValues, j, k);
        }
    }

    hipMemcpy(values, innerValues, size, hipMemcpyDeviceToHost);
    hipFree(innerValues);
}

int main() {
    int *values = (int *) malloc(NUM_VALS * sizeof(int));
    int *origValues = (int *) malloc(NUM_VALS * sizeof(int));
    int n = 50;
    rand_nums(values, n);

    for (int i = 0; i < n; i++) {
        origValues[i] = values[i];
    }

    bitonicSort(values, n);
    hipDeviceSynchronize();
    printf("\nAfter:\n ");
    for (int i = 0; i < n; i++) {
        printf("%d\t|%d\n ", values[i], origValues[i]);
    }
    printf("\n");
}


