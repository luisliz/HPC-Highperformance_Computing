
#include <hip/hip_runtime.h>
#include <stdlib.h> 
#include <stdio.h> 
#include <time.h> 

#define THREADS 512 
#define BLOCKS 32768
#define NUM_VALS THREADS*BLOCKS

void rand_nums(int *values, int length) {
				int i; 
				for(i = 0; i< length; ++i) {
								values[i] = (int)rand()/(int)INT_MAX;
				}
}



__global__ void bitonicMinorSort(int *innerValues, int j, int k) {
				unsigned int i, ixj; 
				i = threadIdx.x + blockDim.x * blockIdx.x; 

				ixj = i^j; 

				if ((ixj)>i) {
								if((i&k)==0) {
												if(innerValues[i]> innerValues[ixj]) {
																int temp = innerValues[i]; 
																innerValues[i] = innerValues[ixj]; 
																innerValues[ixj] = temp; 
												}
								}


								if((i&k)!=0) {
												if(innerValues[i] < innerValues[ixj]) {
																int temp = innerValues[i];
																innerValues[i]	 = innerValues[ixj]; 
																innerValues[ixj] = temp; 
												}
								}
				}
}
void bitonicSort(int *values) {
				int *innerValues	; 

				size_t size = NUM_VALS * sizeof(int); 

				hipMalloc((void **) &innerValues, size); 
				hipMemcpy(innerValues, values, size, hipMemcpyHostToDevice); 

				dim3 blocks(BLOCKS, 1); 
				dim3 threads(THREADS, 1); 

				int j, k; 

				for(k = 2; k <= NUM_VALS; k <<= 1) {
								for(j = k >>1; j>0; j=j>>1) {
												bitonicMinorSort<<<blocks, threads>>>(innerValues, j, k); 
								}
				}

				hipMemcpy(values, innerValues, size, hipMemcpyDeviceToHost); 
				hipFree(innerValues); 
}

int main() {
				int *values = (int *) malloc(NUM_VALS * sizeof(int)); 
				rand_nums(values, NUM_VALS); 
				for(int i = 0; i < NUM_VALS; i++) {
								printf("%d ", values[i]); 
				}
				bitonicSort(values); 

				for(int i = 0; i < NUM_VALS; i++) {
								printf("%d ", values[i]); 
				}
}
