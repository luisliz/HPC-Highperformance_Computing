
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

#define THREADS 1024
#define BLOCKS 65536
#define NUM_VALS THREADS*BLOCKS

void rand_nums(int *values, unsigned long length) {
	int i;
	for (i = 0; i < length; ++i) {
		values[i] = rand() % INT_MAX + 1;;
	}
}


__global__ void bitonicMinorSort(int *innerValues, int j, int k) {
	unsigned int i, ixj;
	i = threadIdx.x + blockDim.x * blockIdx.x;

	ixj = i ^ j;

	if ((ixj) > i) {
		if ((i & k) == 0) {
			if (innerValues[i] > innerValues[ixj]) {
				int temp = innerValues[i];
				innerValues[i] = innerValues[ixj];
				innerValues[ixj] = temp;
			}
		}


		if ((i & k) != 0) {
			if (innerValues[i] < innerValues[ixj]) {
				int temp = innerValues[i];
				innerValues[i] = innerValues[ixj];
				innerValues[ixj] = temp;
			}
		}
	}
}

void bitonicSort(int *values, unsigned long n) {
	int *innerValues;

	size_t size = n * sizeof(int);

	hipMalloc((void **) &innerValues, size);
	hipMemcpy(innerValues, values, size, hipMemcpyHostToDevice);

	int blockSize;
	if(n < THREADS) {
		blockSize = 1;
	} else {
		blockSize = ceil(n/THREADS);
	}

	dim3 blocks(blockSize, 1);
	dim3 threads(THREADS, 1);

	int j, k;

	for (k = 2; k <= n; k <<= 1) {
		for (j = k >> 1; j > 0; j = j >> 1) {
			bitonicMinorSort<<<blocks, threads>>> (innerValues, j, k);
		}
	}

	hipMemcpy(values, innerValues, size, hipMemcpyDeviceToHost);
	hipFree(innerValues);
}

int main(int argc, char *argv[]) {
	int k = 10;  

	if(argc==2)
		k = atoi(argv[1]); 

	int *values = (int *) malloc(NUM_VALS * sizeof(int));    
	int *origValues = (int *) malloc(NUM_VALS * sizeof(int));    
	
	unsigned long n;
	double time_spent;
	clock_t begin, end; 

	for(int i = 0; i < k; i++) {
		n = pow(2,k); 
		printf("\nk = %d, n = %ld\n", k, n); 
		
		rand_nums(values, n);

		for (unsigned long i = 0; i < n; i++)
			origValues[i] = values[i];
		

		time_spent = 0.0; 
		begin = clock(); 
		bitonicSort(values, n);
		end = clock();
		time_spent += (double)(end-begin) / CLOCKS_PER_SEC; 
		printf("\tElapsed time: %f seconds", time_spent); 
	}
}



